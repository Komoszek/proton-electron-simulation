#include "ElectricFieldCuda.hpp"
#include "simulation-cuda.hpp"
#include <random>

ElectricFieldCuda::ElectricFieldCuda(int p_count, int width, int height){
    this->p_count = p_count;
    this->width = width;
    this->height = height;

    int p_size = this->p_count * sizeof(float);

    hipMalloc(&p_x, p_size);
    hipMalloc(&p_y, p_size);
    hipMalloc(&p_c, p_size);
    hipMalloc(&p_m, p_size);

    hipMalloc(&p_Vx, p_size);
    hipMalloc(&p_Vy, p_size);

    hipMalloc(&cIndices, this->p_count*sizeof(int));
    hipMalloc(&pIndices, this->p_count*sizeof(int));

    hipMalloc(&cIndices_start, GRID_SIZE * sizeof(int));
    hipMalloc(&cIndices_end, GRID_SIZE * sizeof(int));

    int texture_size = this->width * this->height * sizeof(float);
    hipMalloc(&potential_x, texture_size);
    hipMalloc(&potential_y, texture_size);

    hipMemset(potential_x, 0, texture_size);
    hipMemset(potential_y, 0, texture_size);


    // int grid_size = GRID_SIZE * sizeof(float);
    
    // WORKAROUND so i don't have to add new variables - have to change
    int grid_size = (GRID_SIZE + MAJOR_GRID_SIZE) * sizeof(float);

    // TODO SPLIT INTO TWO
    hipMalloc(&uniform_grid_p_x, grid_size);
    hipMalloc(&uniform_grid_p_y, grid_size);
    hipMalloc(&uniform_grid_p_c, grid_size);

    hipMalloc(&uniform_grid_n_x, grid_size);
    hipMalloc(&uniform_grid_n_y, grid_size);
    hipMalloc(&uniform_grid_n_c, grid_size);

    #ifdef MEASURE_TIME

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start);

    #endif

    generateParticleData(RANDOM_SEED);

    #ifdef MEASURE_TIME

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    dataGenerateTime += milliseconds;

    #endif
    
    cudaRegisterPBO();
}

void ElectricFieldCuda::generateParticleData(int seed) {
    int p_size = p_count * sizeof(float);
    std::mt19937 gen(RANDOM_SEED);
    std::uniform_real_distribution<> distrPos(0.25f, 0.75f);
    std::uniform_real_distribution<> distrSpeed(-40.0f, 40.0f);
    std::uniform_int_distribution<> distBin(0, 1);

    float * tempArr = new float[p_count];
    float * tempArr2 = new float[p_count];

    for(int i = 0; i < p_count; i++){
        tempArr[i] = distrPos(gen) * width;
    }

    #ifdef MEASURE_TIME

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    hipEventRecord(start);

    #endif

    hipMemcpy(p_x, tempArr, p_size, hipMemcpyHostToDevice);

    #ifdef MEASURE_TIME

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    memcpyTime += milliseconds;

    #endif


    for(int i = 0; i < p_count; i++){
        tempArr[i] = distrPos(gen) * height;
    }

    #ifdef MEASURE_TIME
    
    hipEventRecord(start);

    #endif

    hipMemcpy(p_y, tempArr, p_size, hipMemcpyHostToDevice);

    #ifdef MEASURE_TIME

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    memcpyTime += milliseconds;

    #endif

    for(int i = 0; i < p_count; i++){
        tempArr[i] = distrSpeed(gen);
    }

    #ifdef MEASURE_TIME
    
    hipEventRecord(start);

    #endif

    hipMemcpy(p_Vx, tempArr, p_size, hipMemcpyHostToDevice);

    #ifdef MEASURE_TIME

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    memcpyTime += milliseconds;

    #endif

    for(int i = 0; i < p_count; i++){
        tempArr[i] = distrSpeed(gen);
    }

    #ifdef MEASURE_TIME
    
    hipEventRecord(start);

    #endif

    hipMemcpy(p_Vy, tempArr, p_size, hipMemcpyHostToDevice);

    #ifdef MEASURE_TIME

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    memcpyTime += milliseconds;

    #endif

    for(int i = 0; i < p_count; i++){
        if (distBin(gen)) {
            tempArr[i] = PROTON_CHARGE;
            tempArr2[i] = PROTON_MASS;
        } else {
            tempArr[i] = ELECTRON_CHARGE;
            tempArr2[i] = ELECTRON_MASS;
        }
    }

    #ifdef MEASURE_TIME
    
    hipEventRecord(start);

    #endif

    hipMemcpy(p_c, tempArr, p_size, hipMemcpyHostToDevice);
    hipMemcpy(p_m, tempArr2, p_size, hipMemcpyHostToDevice);

    #ifdef MEASURE_TIME

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    memcpyTime += milliseconds;

    #endif


    delete [] tempArr;
    delete [] tempArr2;
}

ElectricFieldCuda::~ElectricFieldCuda(){
    hipFree(p_x);
    hipFree(p_y);
    hipFree(p_c);
    hipFree(p_m);

    hipFree(p_Vx);
    hipFree(p_Vy);

    hipFree(potential_x);
    hipFree(potential_y);

    hipFree(uniform_grid_p_x);
    hipFree(uniform_grid_p_y);
    hipFree(uniform_grid_p_c);

    hipFree(uniform_grid_n_x);
    hipFree(uniform_grid_n_y);
    hipFree(uniform_grid_n_c);


    hipFree(cIndices);
    hipFree(pIndices);
    hipFree(cIndices_end);
    hipFree(cIndices_start);
}

void ElectricFieldCuda::updateTexture(float dt) {
    cudaUpdateTexture(this, dt);    
}